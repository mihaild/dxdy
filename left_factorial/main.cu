#include <iostream>
#include <vector>
#include <chrono>
#include <algorithm> // For std::min

// CUDA runtime
#include <hip/hip_runtime.h>

#define RESET   "\033[0m"
#define BLUE    "\033[34m"      /* Blue */

// Macro for checking CUDA errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * @brief CUDA kernel to process a block of prime numbers.
 * * Each GPU thread processes one prime number independently. The loop for 'n' for each prime 'p'
 * runs from 2 up to p-1. This is an optimization over the original CPU code, as any
 * calculations for n >= p do not change the final result.
 * * @param p_dev         Device pointer to the array of primes in the current block.
 * @param s_out_dev     Device pointer to store the final 's' value for each prime.
 * @param zeroes_out_dev Device pointer to store the count of 'zeroes' for each prime.
 * @param block_size    The number of primes in the current block.
 */
__global__ void process_block_kernel(const uint64_t* p_dev, uint64_t* s_out_dev, int* zeroes_out_dev, size_t block_size) {
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure we don't process out of bounds
    if (i >= block_size) {
        return;
    }

    // Get the prime number for this thread
    uint64_t p = p_dev[i];

    // Local variables for the calculation
    uint64_t r = 1;
    uint64_t s = 1;
    int zeroes = 0;

    // The main computation loop for the prime 'p'.
    // We calculate s = (1! + 2! + ... + (p-1)!) mod p
    __int128 m = uint64_t(-1) / p;
    for (uint64_t n = 2; n < p; ++n) {
        // r becomes n! mod p
        //r = (r * n) % p;

        r = r * n;
        uint64_t q = (m * r) >> 64;
        uint64_t t = r - q * p;
        r = t - p * (t >= p);
        // s becomes the cumulative sum of factorials mod p
        s += r;
        if (s >= p) {
            s -= p;
        }
        if (s == 0) {
            zeroes++;
        }
    }

    // Store the final results back into global memory
    s_out_dev[i] = s;
    zeroes_out_dev[i] = zeroes;
}


int main() {
    uint64_t N;
    std::cout << "Enter the upper limit N to find primes: ";
    std::cin >> N;

    // --- Step 1: Sieve of Eratosthenes to find primes on the CPU ---
    std::vector<bool> is_prime(N, true);
    std::vector<uint64_t> primes;
    is_prime[0] = is_prime[1] = false;
    for (int i = 2; i < N; ++i) {
        if (is_prime[i]) {
            primes.push_back(i);
            for (int j = i + i; j < N; j += i) {
                is_prime[j] = false;
            }
        }
    }

    if (primes.empty()) {
        std::cerr << "No primes found up to " << N << std::endl;
        return 1;
    }
    std::cerr << "Will check " << primes.size() << " primes, up to " << primes.back() << std::endl;

    // --- Step 2: Set up for block processing and CUDA ---
    auto start = std::chrono::system_clock::now();
    auto last = start;

    // Size of blocks to process on the GPU at a time. Tune for performance.
    constexpr size_t BLOCK_SIZE = 1024 * 128;

    // For progress reporting
    std::vector<uint64_t> cum_sums(primes);
    for (size_t i = 1; i < cum_sums.size(); ++i) {
        cum_sums[i] += cum_sums[i - 1];
    }

    // Host vectors to hold results from the GPU for one block
    std::vector<uint64_t> s_host(BLOCK_SIZE);
    std::vector<int> zeroes_host(BLOCK_SIZE);

    // Device pointers
    uint64_t* p_dev = nullptr;
    uint64_t* s_dev = nullptr;
    int* zeroes_dev = nullptr;

    // Allocate memory on the GPU for one block
    gpuErrchk(hipMalloc(&p_dev, BLOCK_SIZE * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&s_dev, BLOCK_SIZE * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&zeroes_dev, BLOCK_SIZE * sizeof(int)));

    // --- Step 3: Main loop to process primes in blocks ---
    for (size_t block_start = 0; block_start < primes.size(); block_start += BLOCK_SIZE) {
        // Progress reporting (similar to the original)
        auto cur = std::chrono::system_clock::now();
        if (std::chrono::duration<double>(cur - last).count() > 10) {
            double t = std::chrono::duration<double>(cur - start).count();
            double expected_total = t / cum_sums[block_start] * cum_sums.back();
            std::cerr << BLUE << "i=" << block_start << ", t=" << t << ", expected_total=" << expected_total << RESET << std::endl;
            last = cur;
        }

        const size_t current_block_size = std::min(BLOCK_SIZE, primes.size() - block_start);

        // Copy current block of primes from host to device
        gpuErrchk(hipMemcpy(p_dev, primes.data() + block_start, current_block_size * sizeof(uint64_t), hipMemcpyHostToDevice));

        // Configure and launch the CUDA kernel
        const int threads_per_block = 256;
        const int blocks_per_grid = (current_block_size + threads_per_block - 1) / threads_per_block;
        process_block_kernel<<<blocks_per_grid, threads_per_block>>>(p_dev, s_dev, zeroes_dev, current_block_size);

        // Check for any errors during kernel launch
        gpuErrchk(hipGetLastError());
        // Wait for the kernel to complete
        gpuErrchk(hipDeviceSynchronize());

        // Copy results from device back to host
        gpuErrchk(hipMemcpy(s_host.data(), s_dev, current_block_size * sizeof(uint64_t), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(zeroes_host.data(), zeroes_dev, current_block_size * sizeof(int), hipMemcpyDeviceToHost));

        // Process results on the CPU
        for (size_t i = 0; i < current_block_size; ++i) {
            if (zeroes_host[i] >= 8 || s_host[i] == 0) {
                std::cout << primes[i + block_start] << ' ' << (s_host[i] == 0) << ' ' << zeroes_host[i] << std::endl;
            }
        }
    }

    // --- Step 4: Cleanup ---
    gpuErrchk(hipFree(p_dev));
    gpuErrchk(hipFree(s_dev));
    gpuErrchk(hipFree(zeroes_dev));

    std::cout << "total time " << std::chrono::duration<double>(std::chrono::system_clock::now() - start).count() << std::endl;
    return 0;
}

